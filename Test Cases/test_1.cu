/**
 * Naive Example of Matrix Addition
 *
 */

/**
 * Matrix multiplication: C = A + B.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>


void constantInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
      data[i] = (float)rand()/RAND_MAX;
    }
}

int matrixAdd_gold(float *A, float *B, float*C, int size){
  for (int i=0;i<size;i++)
    C[i] = A[i] + B[i];
  return 0;
}

/**
 * Matrix addition (CUDA Kernel) on the device: C = A + B
 * w is matrix width, h is matrix height
 */
__global__ void
matrixAddCUDA(float *C, float *A, float *B, int w, int h)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    printf("Referrence of blockIdx.x: %d.\n", blockIdx.x);
    printf("Referrence of blockIdx.y: %d.\n", blockIdx.y);

    // Thread local index
    int txl = threadIdx.x;
    int tyl = threadIdx.y;

    // Thread global index
    int tx = txl+bx*blockDim.x;
    int ty = tyl+by*blockDim.y;
    int glbIdx = ty*w+tx;

    int maxidx = w*h-1;
    if (glbIdx<0 || glbIdx>maxidx){
      printf("Error: glbIdx is %d.\n", glbIdx);
    }
    else{
      // Do addition
      C[glbIdx] = A[glbIdx] + B[glbIdx];
    }
    // if (threadIdx.x==0 && threadIdx.y==0){
    //   printf("bx=%d, by=%d, txl=%d, tyl=%d, glbIdx=%d, A[glbIdx]=%f, B[glbIdx]=%f, C[glbIdx]=%f\n",
    // 	     bx, by, txl, tyl, glbIdx, A[glbIdx], B[glbIdx], C[glbIdx]);
    // }
}

/**
 * A wrapper that calls the GPU kernel
 */
int matrixAdd(int block_size, int w, int h)
{
    // Allocate host memory for matrices A and B
  unsigned int sz = w*h;
  unsigned int mem_size = sizeof(float) * sz;
  float *h_A = (float *)malloc(mem_size);
  float *h_B = (float *)malloc(mem_size);
  float *h_C = (float *) malloc(mem_size);
  
    // Initialize host memory
    constantInit(h_A, sz);
    constantInit(h_B, sz);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipError_t error;
    error = hipMalloc((void **) &d_A, mem_size);
    error = hipMalloc((void **) &d_B, mem_size);
    error = hipMalloc((void **) &d_C, mem_size);
    
    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(w / threads.x, h / threads.y);
    dim3 newGrid(w / threads.x, h / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    matrixAddCUDA<<< newGrid, threads >>>(d_C, d_A, d_B, w, h);

    printf("done\n");

    hipDeviceSynchronize();

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    /* check the result correctness */
    float g_sum=0, c_sum=0;
    for (int i=0;i<w*h;i++)      {
      //      if (fmod(i,32*w)==0) printf("h_C[%d]=%f\n",i,h_C[i]);
      g_sum += h_C[i];
    }
    matrixAdd_gold(h_A, h_B, h_C, w*h);
    for (int i=0;i<w*h;i++)       c_sum += h_C[i];    
    if (abs(g_sum - c_sum)<1e-10){
      printf("Pass...\n");
    }
    else{
      printf("Fail: %f vs. %f.\n", g_sum, c_sum);
    }

    matrixAddCUDA<<< newGrid, threads >>>(d_C, d_A, d_B, w, h);

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

